#include "hip/hip_runtime.h"
#include "ai_data.h"
#include "ai_methods.h"
#include "simulator_data.h"
#include "simulator_methods.h"
#include "thread_data.h"
#include "thread_methods.h"
#include "utils.h"

// hiprand on device
#include <hiprand/hiprand_kernel.h>

// for debug purposes
#include <iostream>


/*  startFunc<<<1, 1>>>(start_params);
    while (handle) {
        run out_graph;
        updateFunc<<<1, 1>>>(update_args);
    }
*/
hipGraph_t cudaGraphWhile(hipGraph_t graph, cudaGraphConditionalHandle handle,
    void* startFunc, void* start_args[], void* updateFunc, void* update_args[]) {

    hipGraphNodeParams start_params = { hipGraphNodeTypeKernel };
    start_params.kernel.func = (void*) startFunc;
    start_params.kernel.gridDim = dim3(1, 1, 1);
    start_params.kernel.blockDim = dim3(1, 1, 1);
    start_params.kernel.kernelParams = start_args;
    hipGraphNode_t start_node;
    hipGraphAddNode(&start_node, graph, NULL, 0, &start_params);

    hipGraphNodeParams cond_params = { hipGraphNodeTypeConditional };
    cond_params.conditional.handle = handle;
    cond_params.conditional.type = cudaGraphCondTypeWhile;
    cond_params.conditional.size = 1;
    hipGraphNode_t cond_node;
    hipGraphAddNode(&cond_node, graph, {&start_node}, 1, &cond_params);

    hipGraph_t body_graph = condParams.conditional.phGraph_out[0];

    hipGraphNode_t out_node;
    hipGraphAddEmptyNode(&out_node, body_graph, NULL, 0);
    hipGraph_t out_graph;
    hipGraphCreate(&out_graph, 0);
    hipGraphChildGraphNodeGetGraph(out_node, &out_graph);

    hipGraphNodeParams update_params = { hipGraphNodeTypeKernel };
    update_params.kernel.func = (void*) updateFunc;
    update_params.kernel.gridDim = dim3(1, 1, 1);
    update_params.kernel.blockDim = dim3(1, 1, 1);
    update_params.kernel.kernelParams = update_args;
    hipGraphNode_t update_node;
    hipGraphAddNode(&update_node, body_graph, {&out_node}, 1, &update_params);

    return out_graph;
}

__global__ void whileTimeStart(cudaGraphConditionalHandle handle, ull* finish_time, ull time) {
    *finish_time = Utils::get_globaltimer() + time;
    cudaGraphSetConditional(handle, 1);
}

__global__ void whileTimeUpdate(cudaGraphConditionalHandle handle, ull* finish_time) {
    cudaGraphSetConditional(handle, Utils::get_globaltimer() < *finish_time ? 1 : 0);
}

// while globaltime < start + time: run subgraph
// return subgraph, input time in ns
hipGraph_t whileTime(ull time, hipGraph_t graph) {
    cudaGraphConditionalHandle handle;
    cudaGraphConditionalHandleCreate(&handle, graph);
    volatile ull* finish_time;
    hipMalloc(&finish_time, sizeof(ull));
    return cudaGraphWhile(graph, handle,
        (void*) whileTimeStart, (void*[3]){&handle, &finish_time, &time},
        (void*) whileTimeUpdate, (void*[2]){&handle, &finish_time});
}


/*  while work time < [work_time] s:
        forward step
        simulator step
        backward step
        if fin: reset simulator
*/
void runAll() {
    // user:: fill train parameters
    const int threads_num = 10;
    const int batch_size = 100;
    const int work_time = 30; // seconds

    auto [data, size] = init(threads_num);

    hipGraph_t graph;
    hipGraphCreate(&graph, 0);

    hipGraph_t empty_graph;
    hipGraphCreate(&empty_graph, 0);

    for (uint thread_id = 0; thread_id < threads_num; ++thread_id) {

        hipGraph_t body_graph = whileTime(work_time * (ull) 1e9, graph);

        // forward step
        hipGraphNode_t forward_node;
        hipGraphAddChildGraphNode(&forward_node, body_graph, NULL, 0, empty_graph);
        hipGraph_t forward_graph;
        hipGraphChildGraphNodeGetGraph(forward_node, &forward_graph);
        AI::forwardStep(&forward_graph, data[thread_id]);
        
        hipGraphDestroy(forward_graph);

        // simulator step
        hipGraphNode_t simulator_node;
        hipGraphAddChildGraphNode(&simulator_node, body_graph, {&forward_node}, 1, empty_graph);
        hipGraph_t simulator_graph;
        hipGraphChildGraphNodeGetGraph(simulator_node, &simulator_graph);
        Simulator::stepReset(&simulator_graph, data[thread_id]->simulator, size, reset_handle);

        // backward step
        hipGraphNode_t backward_node;
        hipGraphAddChildGraphNode(&backward_node, body_graph, {&simulator_node}, 1, empty_graph);
        hipGraph_t backward_graph;
        hipGraphChildGraphNodeGetGraph(backward_node, &backward_graph);
        AI::backwardStep(&backward_graph, data[thread_id]);

        // if reached end of simulation: reset simulator
        hipGraphNodeParams reset_cond_params = { hipGraphNodeTypeConditional };
        reset_cond_params.conditional.handle = reset_handle;
        update_params.kernel.type = cudaGraphCondTypeIf;
        update_params.kernel.size = 1;
        hipGraphNode_t reset_cond_node;
        hipGraphAddNode(&reset_cond_node, body_graph, {&backward_node}, 1, &reset_cond_params);
        hipGraph_t reset_cond_graph = reset_cond_params.conditional.phGraph_out[0];

        hipGraphNodeParams reset_params = { hipGraphNodeTypeKernel };
        void* reset_args[] = {&data, &data_base};
        reset_params.kernel.func = (void*) Simulator::copyDeviceToDevice;
        reset_params.kernel.gridDim = dim3(1);
        reset_params.kernel.blockDim = dim3(1);
        reset_params.kernel.kernelParams = reset_args;
        hipGraphNode_t reset_node;
        hipGraphAddNode(&reset_node, reset_cond_graph, NULL, 0, &reset_params);
    }

    hipGraphExec_t graph_exec;

    hipGraphInstantiate(&graph_exec, graph, NULL, NULL, 0);
    hipGraphLaunch(graph_exec, 0);

    hipDeviceSynchronize();

    hipGraphExecDestroy(graph_exec);
    hipGraphDestroy(graph);

    // write weights to system

    // there is some data allocated in host and device memory
    // should be cleaned by OS
}


int main() {

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);

    runAll();

    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipDeviceSynchronize();

    float ms = 0;
    hipEventElapsedTime(&ms, start, stop);
    std::cout << "CUDA time (ms): " << ms << std::endl;

    std::cout << hipGetErrorString(hipGetLastError()) << std::endl;
}