#include "hip/hip_runtime.h"
#include "thread_data.h"
#include "thread_methods.h"

#include "character.h"

#include "simulator_data.h"
#include "simulator_methods.h"

#include "utils.h"

#include "lodepng.h"
#include "heatmap.h"
#include "gif.h"

#include "colorschemes/Spectral.h"
#include "colorschemes/gray.h"

#include <iostream>
#include <string>
#include <fstream>
#include <filesystem>

using Character::Agent;
using Character::Player;
using Character::Astral;
using Character::Cultist;

using PAct = Player::Action;
using AAct = Astral::Action;
using CAct = Cultist::Action;

void render_heatmap(Simulator::Data* data, float (*f) (Simulator::Data*, uint), std::string name, GifWriter* gif = NULL, const heatmap_colorscheme_t* colorscheme = heatmap_cs_Spectral_discrete) {
    
    heatmap_t* hm = heatmap_new(data->width, data->height);
    heatmap_stamp_t* stamp = heatmap_stamp_gen(0);
    for(uint y = 0; y < data->height; ++y) {
        for (uint x = 0; x < data->width; ++x) {
            float res = f(data, y * data->width + x);
            heatmap_add_weighted_point_with_stamp(hm, x, y, res == 0 ? 0.01 : res, stamp);
        }
    }

    std::vector<unsigned char> image(4 * data->height * data->width);
    heatmap_render_saturated_to(hm, colorscheme, 1.f, &image[0]);

    std::vector<unsigned char> image_rev(4 * data->height * data->width);
    for (uint y = 0; y < data->height; ++y) {
        for (uint x = 0; x < data->width; ++x) {
            for (uint c = 0; c < 4; ++c) {
                image_rev[4 * data->width * y + 4 * x + c] = image[4 * data->width * (data->height - y - 1) + 4 * x + c];
            }
        }
    }

    heatmap_free(hm);

    unsigned err = lodepng::encode(name + ".png", image_rev, data->width, data->height);

    if (gif != NULL) {
        for (uint y = 0; y < data->height; ++y) {
            for (uint x = 0; x < data->width; ++x) {
                uint ind = 4 * (data->width * y + x); 
                if (image_rev[ind + 3] == 0) {
                    image_rev[ind] = (unsigned char) 255;
                    image_rev[ind + 1] = (unsigned char) 255;
                    image_rev[ind + 2] = (unsigned char) 255;
                }
            }
        }
        GifWriteFrame(gif, image_rev.data(), data->width, data->height, 0);
    }
}

void init_data_heatmap(float (*f) (Simulator::Data*, uint), std::string name) {
    Simulator::Data* data = Simulator::initHost();
    render_heatmap(data, f, name);
    Simulator::deleteHost(data);
}

void copy_heatmap(float (*f) (Simulator::Data*, uint), std::string name) {
    Simulator::Data* data = Simulator::initHost();
    Simulator::Data* data_device = Simulator::copyHostToDevice(data);
    Simulator::deleteHost(data);
    data = Simulator::copyDeviceToHost(data_device);
    render_heatmap(data, f, name);
    Simulator::deleteHost(data);
    Simulator::deleteDevice(data_device);
}

float object_map(Simulator::Data* data, uint i) {
    return (float) data->object_map[i] / (uint) Simulator::ObjectType::Last;
}

float player_sight_map(Simulator::Data* data, uint i) {
    return (float) *data->player_tensor.getSight(i);
}

float monsters_sight_map(Simulator::Data* data, uint i) {
    return (float) *data->monsters_tensor.getSight(i);
}

float player_hearing_map(Simulator::Data* data, uint i) {
    return (float) *(data->player_tensor.getHearing(i));
}

float monsters_hearing_map(Simulator::Data* data, uint i) {
    return (float) *(data->monsters_tensor.getHearing(i));
}

float lighting_map(Simulator::Data* data, uint i) {
    return (float) max((uint) (
            data->madness_world ?
            data->lighting_map.madness[i] :
            data->lighting_map.normal[i]),
        (uint) data->lighting_map.dynamic[i]) / 2.f;
}

void write_agents(Simulator::Data* data, std::string name) {
    std::ofstream player_file;
    player_file.open("./out/" + name + "/_agents/player.csv", std::ios_base::app);
    half* player_data = data->agents[0].agent->data;
    for (uint i = 0; i < Player::data_size - 1; ++i) {
        player_file << std::to_string(__half2float(player_data[i])) << ", ";
    }
    player_file << std::to_string(__half2float(player_data[Player::data_size - 1])) << std::endl;
    player_file.close();

    for (uint astral_id = 0; astral_id < 2; ++astral_id) {
        std::ofstream astral_file;
        astral_file.open("./out/" + name + "/_agents/astral_" + std::to_string(astral_id + 1) + ".csv", std::ios_base::app);
        half* astral_data = data->agents[1 + astral_id].agent->data;
        for (uint i = 0; i < Astral::data_size - 1; ++i) {
            astral_file << std::to_string(__half2float(astral_data[i])) << ", ";
        }
        astral_file << std::to_string(__half2float(astral_data[Astral::data_size - 1])) << std::endl;
        astral_file.close();
    }

    for (uint cultist_id = 0; cultist_id < 2; ++cultist_id) {
        std::ofstream cultist_file;
        cultist_file.open("./out/" + name + "/_agents/cultist_" + std::to_string(cultist_id + 1) + ".csv", std::ios_base::app);
        half* cultist_data = data->agents[3 + cultist_id].agent->data;
        for (uint i = 0; i < Cultist::data_size - 1; ++i) {
            cultist_file << std::to_string(__half2float(cultist_data[i])) << ", ";
        }
        cultist_file << std::to_string(__half2float(cultist_data[Cultist::data_size - 1])) << std::endl;
        cultist_file.close();
    }
    
}

void clear_agents(std::string name) {
    std::ofstream file;
    std::filesystem::remove_all("./out/" + name + "/");
    std::filesystem::create_directory("./out/");
    std::filesystem::create_directory("./out/" + name + "/");
    std::filesystem::create_directory("./out/" + name + "/_agents/");
    std::ofstream {"./out/" + name + "/_agents/player.csv"};
    file.open("./out/" + name + "/_agents/player.csv", std::ofstream::out | std::ofstream::trunc);
    file << "x_int, x_frac, y_int, y_frac, rot_sin, rot_cos, cooldown, madness, health, " <<
            "sight_rad, sound_rad, hearing_rad, mov_state, speed, mag_see, mag_inv, mag_ina, "
            "noise, door, window, stamina, shadow_x, shadow_y" << std::endl;
    file.close();
    std::ofstream {"./out/" + name + "/_agents/astral_1.csv"};
    file.open("./out/" + name + "/_agents/astral_1.csv", std::ofstream::out | std::ofstream::trunc);
    file << "x_int, x_frac, y_int, y_frac, rot_sin, rot_cos, cooldown, madness, health, " <<
            "speed, sound_rad, abil_cooldown, scream_cooldown, see, attack" << std::endl;
    file.close();
    std::ofstream {"./out/" + name + "/_agents/astral_2.csv"};
    file.open("./out/" + name + "/_agents/astral_2.csv", std::ofstream::out | std::ofstream::trunc);
    file << "x_int, x_frac, y_int, y_frac, rot_sin, rot_cos, cooldown, madness, health, " <<
            "speed, sound_rad, abil_cooldown, scream_cooldown, see, attack" << std::endl;
    file.close();
    std::ofstream {"./out/" + name + "/_agents/cultist_1.csv"};
    file.open("./out/" + name + "/_agents/cultist_1.csv", std::ofstream::out | std::ofstream::trunc);
    file << "x_int, x_frac, y_int, y_frac, rot_sin, rot_cos, cooldown, madness, health, " <<
            "noise, sound_rad, abil_cooldown" << std::endl;
    file.close();
    std::ofstream {"./out/" + name + "/_agents/cultist_2.csv"};
    file.open("./out/" + name + "/_agents/cultist_2.csv", std::ofstream::out | std::ofstream::trunc);
    file << "x_int, x_frac, y_int, y_frac, rot_sin, rot_cos, cooldown, madness, health, " <<
            "noise, sound_rad, abil_cooldown" << std::endl;
    file.close();
}

void log_data(Simulator::Data* data, uint action_id, std::string name, GifWriter* gifs) {
    std::filesystem::create_directory("./out/" + name + "/" + std::to_string(action_id) + "/");

    std::filesystem::create_directory("./out/" + name + "/" + std::to_string(action_id) + "/player/");
    render_heatmap(data,
        [](Simulator::Data* data, uint i) {return __half2float(*data->player_tensor.getObject(i));},
        "./out/" + name + "/" + std::to_string(action_id) + "/player/object_map", gifs);
    render_heatmap(data,
        [](Simulator::Data* data, uint i) {return __half2float(*data->player_tensor.getLighting(i));},
        "./out/" + name + "/" + std::to_string(action_id) + "/player/lighting_map", gifs + 1, heatmap_cs_w2b);
    render_heatmap(data,
        [](Simulator::Data* data, uint i) {return __half2float(*data->player_tensor.getSight(i));},
        "./out/" + name + "/" + std::to_string(action_id) + "/player/sight_map", gifs + 2, heatmap_cs_w2b);
    render_heatmap(data,
        [](Simulator::Data* data, uint i) {return __half2float(*data->player_tensor.getSound(i));},
        "./out/" + name + "/" + std::to_string(action_id) + "/player/sound_map", gifs + 3, heatmap_cs_w2b);
    render_heatmap(data,
        [](Simulator::Data* data, uint i) {return __half2float(*data->player_tensor.getHearing(i));},
        "./out/" + name + "/" + std::to_string(action_id) + "/player/hearing_map", gifs + 4, heatmap_cs_w2b);

    std::filesystem::create_directory("./out/" + name + "/" + std::to_string(action_id) + "/monsters/");
    render_heatmap(data,
        [](Simulator::Data* data, uint i) {return __half2float(*data->monsters_tensor.getObject(i));},
        "./out/" + name + "/" + std::to_string(action_id) + "/monsters/object_map", gifs + 5);
    render_heatmap(data,
        [](Simulator::Data* data, uint i) {return __half2float(*data->monsters_tensor.getLighting(i));},
        "./out/" + name + "/" + std::to_string(action_id) + "/monsters/lighting_map", gifs + 6, heatmap_cs_w2b);
    render_heatmap(data,
        [](Simulator::Data* data, uint i) {return __half2float(*data->monsters_tensor.getSight(i));},
        "./out/" + name + "/" + std::to_string(action_id) + "/monsters/sight_map", gifs + 7, heatmap_cs_w2b);
    render_heatmap(data,
        [](Simulator::Data* data, uint i) {return __half2float(*data->monsters_tensor.getSound(i));},
        "./out/" + name + "/" + std::to_string(action_id) + "/monsters/sound_map", gifs + 8, heatmap_cs_w2b);
    render_heatmap(data,
        [](Simulator::Data* data, uint i) {return __half2float(*data->monsters_tensor.getHearing(i));},
        "./out/" + name + "/" + std::to_string(action_id) + "/monsters/hearing_map", gifs + 9, heatmap_cs_w2b);
    
    std::filesystem::create_directory("./out/" + name + "/" + std::to_string(action_id) + "/common/");
    render_heatmap(data,
        [](Simulator::Data* data, uint i) {return (float) data->object_map[i] / (uint) Simulator::ObjectType::Last;},
        "./out/" + name + "/" + std::to_string(action_id) + "/common/object_map", gifs + 10);
    render_heatmap(data,
        [](Simulator::Data* data, uint i) {return (float) max(
            (uint) (data->madness_world ? data->lighting_map.madness[i] : data->lighting_map.normal[i]),
            (uint) data->lighting_map.dynamic[i]) / 2.f;},
        "./out/" + name + "/" + std::to_string(action_id) + "/common/lighting_map", gifs + 11, heatmap_cs_w2b);
    
    write_agents(data, name);
}

GifWriter* initGifs(std::string name, uint delay = 0) {
    uint width = 300;
    uint height = 150;
    GifWriter* gifs = (GifWriter*) malloc(12 * sizeof(GifWriter));
    std::filesystem::create_directory("./out/" + name + "/_player/");
    GifBegin(gifs, ("./out/" + name + "/_player/object_map.gif").c_str(), width, height, delay);
    GifBegin(gifs + 1, ("./out/" + name + "/_player/lighting_map.gif").c_str(), width, height, delay);
    GifBegin(gifs + 2, ("./out/" + name + "/_player/sight_map.gif").c_str(), width, height, delay);
    GifBegin(gifs + 3, ("./out/" + name + "/_player/sound_map.gif").c_str(), width, height, delay);
    GifBegin(gifs + 4, ("./out/" + name + "/_player/hearing_map.gif").c_str(), width, height, delay);
    std::filesystem::create_directory("./out/" + name + "/_monsters/");
    GifBegin(gifs + 5, ("./out/" + name + "/_monsters/object_map.gif").c_str(), width, height, delay);
    GifBegin(gifs + 6, ("./out/" + name + "/_monsters/lighting_map.gif").c_str(), width, height, delay);
    GifBegin(gifs + 7, ("./out/" + name + "/_monsters/sight_map.gif").c_str(), width, height, delay);
    GifBegin(gifs + 8, ("./out/" + name + "/_monsters/sound_map.gif").c_str(), width, height, delay);
    GifBegin(gifs + 9, ("./out/" + name + "/_monsters/hearing_map.gif").c_str(), width, height, delay);
    std::filesystem::create_directory("./out/" + name + "/_common/");
    GifBegin(gifs + 10, ("./out/" + name + "/_common/object_map.gif").c_str(), width, height, delay);
    GifBegin(gifs + 11, ("./out/" + name + "/_common/lighting_map.gif").c_str(), width, height, delay);
    return gifs;
}

void endGifs(GifWriter* gifs, std::string name) {
    for (uint i = 0; i < 12; ++i) {
        GifEnd(gifs + i);
    }
}

void test(std::string name, bool allow_death) {
    clear_agents(name);
    Simulator::Data* data = Simulator::initHost();
    Simulator::Data* data_device = Simulator::copyHostToDevice(data);
    Simulator::Data* data_device_base = Simulator::copyHostToDevice(data);
    Simulator::Data data_local;
    hipMemcpy(&data_local, data_device, sizeof(Simulator::Data), hipMemcpyDeviceToHost);
    Simulator::AgentData agents_data[5];
    hipMemcpy(agents_data, data_local.agents, 5 * sizeof(Simulator::AgentData), hipMemcpyDeviceToHost);
    uint* actions_p[5];
    Character::Player* player = (Character::Player*) data->agents[0].agent;
    uint actions_delta = (uint8_t*) (&(player->action)) - (uint8_t*) player;
    for (uint i = 0; i < 5; ++i) {
        actions_p[i] = (uint*) ((uint8_t*) (agents_data[i].agent) + actions_delta);
    }

    GifWriter* gifs = initGifs(name);

    log_data(data, 0, name, gifs);
    Simulator::deleteHost(data);

    std::ifstream actions_file("./tests/" + name + ".actions");
    uint p, a1, a2, c1, c2;
    uint action_id = 0;
    while (actions_file >> p >> a1 >> a2 >> c1 >> c2) {
        ++action_id;

        hipMemcpy(actions_p[0], &p, sizeof(uint), hipMemcpyHostToDevice);
        hipMemcpy(actions_p[1], &a1, sizeof(uint), hipMemcpyHostToDevice);
        hipMemcpy(actions_p[2], &a2, sizeof(uint), hipMemcpyHostToDevice);
        hipMemcpy(actions_p[3], &c1, sizeof(uint), hipMemcpyHostToDevice);
        hipMemcpy(actions_p[4], &c2, sizeof(uint), hipMemcpyHostToDevice);

        hipGraph_t graph;
        hipGraphCreate(&graph, 0);

        if (allow_death) {
            Simulator::stepReset(&graph, data_device, data_device_base, 150 * 300);
        } else {
            Simulator::step(&graph, data_device, 150 * 300);
        }

        hipGraphExec_t graph_exec;
        hipGraphInstantiate(&graph_exec, graph, NULL, NULL, 0);
        
        hipGraphLaunch(graph_exec, 0);
        hipDeviceSynchronize();

        hipGraphExecDestroy(graph_exec);
        hipGraphDestroy(graph);

        data = Simulator::copyDeviceToHost(data_device);

        if (action_id % 10 == 0) {
            std::cout << action_id << std::endl;
        }

        log_data(data, action_id, name, gifs);

        Simulator::deleteHost(data);
    }

    deleteDevice(data_device);
    deleteDevice(data_device_base);

    endGifs(gifs, name);

}

__host__ void test_time(std::string name) {
    Simulator::Data* data = Simulator::initHost();
    Simulator::Data* data_device = Simulator::copyHostToDevice(data);
    Simulator::Data* data_device_base = Simulator::copyHostToDevice(data);
    Simulator::Data data_local;
    hipMemcpy(&data_local, data_device, sizeof(Simulator::Data), hipMemcpyDeviceToHost);
    Simulator::AgentData agents_data[5];
    hipMemcpy(agents_data, data_local.agents, 5 * sizeof(Simulator::AgentData), hipMemcpyDeviceToHost);
    uint* actions_p[5];
    Character::Player* player = (Character::Player*) data->agents[0].agent;
    uint actions_delta = (uint8_t*) (&(player->action)) - (uint8_t*) player;
    for (uint i = 0; i < 5; ++i) {
        actions_p[i] = (uint*) ((uint8_t*) (agents_data[i].agent) + actions_delta);
    }

    Simulator::deleteHost(data);

    std::ifstream actions_file("./tests/" + name + ".actions");
    uint p, a1, a2, c1, c2;
    uint action_id = 0;
    while (actions_file >> p >> a1 >> a2 >> c1 >> c2) {
        ++action_id;

        hipMemcpy(actions_p[0], &p, sizeof(uint), hipMemcpyHostToDevice);
        hipMemcpy(actions_p[1], &a1, sizeof(uint), hipMemcpyHostToDevice);
        hipMemcpy(actions_p[2], &a2, sizeof(uint), hipMemcpyHostToDevice);
        hipMemcpy(actions_p[3], &c1, sizeof(uint), hipMemcpyHostToDevice);
        hipMemcpy(actions_p[4], &c2, sizeof(uint), hipMemcpyHostToDevice);

        hipGraph_t graph;
        hipGraphCreate(&graph, 0);

        Simulator::step(&graph, data_device, 150 * 300);

        hipGraphExec_t graph_exec;
        hipGraphInstantiate(&graph_exec, graph, NULL, NULL, 0);
        
        hipGraphLaunch(graph_exec, 0);
        hipDeviceSynchronize();

        hipGraphExecDestroy(graph_exec);
        hipGraphDestroy(graph);

        data = Simulator::copyDeviceToHost(data_device);

        Simulator::deleteHost(data);
    }

    deleteDevice(data_device);
    deleteDevice(data_device_base);
}

__host__ void test_base() {
    Simulator::Data* data = Simulator::initHost();
    Simulator::Data* data_device = Simulator::copyHostToDevice(data);
    Simulator::deleteHost(data);
    data = Simulator::copyDeviceToHost(data_device);
    Simulator::deleteDevice(data_device);
    data_device = Simulator::copyHostToDevice(data);
    Simulator::Data* data_device_2 = Simulator::copyHostToDevice(data);
    Simulator::deleteHost(data);
    copyDeviceToDevice<<<1, 1>>>(data_device, data_device_2);
    hipDeviceSynchronize();
    Simulator::deleteDevice(data_device_2);
    data = Simulator::copyDeviceToHost(data_device);
    Simulator::deleteDevice(data_device);
    std::filesystem::remove_all("./out/base/");
    std::filesystem::create_directory("./out/base/");
    render_heatmap(data, object_map, "./out/base/object_map");
    render_heatmap(data, player_sight_map, "./out/base/player_sight_map");
    render_heatmap(data, monsters_sight_map, "./out/base/monsters_sight_map");
    render_heatmap(data, player_hearing_map, "./out/base/player_hearing_map");
    render_heatmap(data, monsters_hearing_map, "./out/base/monsters_hearing_map");
    render_heatmap(data, lighting_map, "./out/base/lighting_map");
    Simulator::deleteHost(data);
}


int main(int argc, char* argv[]) {

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);

    if (argc == 1) {
        test_base();
    } else if (argc == 2) {
        std::string name {argv[1]};
        test(name, false);
    } else if (argc == 3) {
        std::string name {argv[1]};
        std::string arg {argv[2]};
        if (arg == "allow-death") {
            test(name, true);
        } else if (arg == "time") {
            test_time(name);
        } else {
            std::cout << "invalid arguments" << std::endl;
        }
    }

    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipDeviceSynchronize();

    float ms = 0;
    hipEventElapsedTime(&ms, start, stop);
    std::cout << "Time (ms): " << ms << std::endl;

    std::cout << hipGetErrorString(hipGetLastError()) << std::endl;
}